#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <math.h>
#include <time.h>
#include <stdlib.h>
#include <fstream>
#include <time.h>
#include "IC_mod.h"


using namespace std;

hipError_t addWithCuda(
    double* T,
    double* pl,
    double* pe,
    double* met,
    double* gam_V,
    double* gam_1,
    double* gam_2,
    double* Beta_V,
    double* Beta,
    double* net_F,
    double* mu_s,
    double* F0,
    double* Fint,
    double* grav,
    double* AB,
    double* cp_air,
    double* kappa_air,
    double* t_step,
    int* n_step,
    double* host_k_IR_l,
    double* host_k_V_l,
    double* dT_rad,
    double* dT_conv,
    unsigned int nlay,
    unsigned int ncol
);



int main()
{
    

    const double StBC = 5.670374419e-8;


    // Number of columns, layers and edges
    const int ncol = 2;
    const int nlay = 52;
    const int nlay1 = nlay + 1;



    const std::string frmat = "(I6.6)";

    // Read in sigma hybrid grid values
    const std::string a_sh = "sig_hyb_HJ_53_a.txt";
    const std::string b_sh = "sig_hyb_HJ_53_b.txt";

    // Surface pressure (pa)
    const double p0 = 2.2e7;

    // step in seconds
    double t_step[ncol] = { 30.0 };

    // Number of steps
    double n_step[ncol] = { 1e5 };

    // Constants
    //double R = 8.31446261815324; //     ! Ideal gas constant
    double Rd_air[ncol]; //          ! Specific gas constant
    double cp_air[ncol]; //           ! Rd_air/kappa_air ! Heat capacity of air
    double kappa_air[ncol];// ! kappa = Rd/cp
    double grav[ncol]; //              ! Gravity
    double met[ncol];   //                ! Metallicity in dex solar, solar[M/H] = 0.0


    //! short wave Zenith angle
    double mu_s[ncol];

    double Tirr[ncol]; //               ! Irradiation temperature
    double Tint[ncol]; //                 ! Internal temperature

    double F0[ncol];   //        ! Substellar point irradiation flux

    double Fint[ncol];//      ! Internal flux

    double k_IR[ncol];        //           ! Constant IR opacity
    double k_V[ncol];//! Constant V opacity

    double gam[ncol]; //               ! Gamma ratio

    // Parmentier IC and parameters
    int iIC = 4; //       ! IC choice here
    bool corr = true; //  ! Do adibatic correction

    int table_num = 1; // ! Table 1 = with TiO/VO, Table 2 = without TiO/VO

    for (size_t i = 0; i < ncol; i++)
    {
        Rd_air[i] = 3556.8;
        cp_air[i] = 1.3e4;
        kappa_air[i] = Rd_air[i] / cp_air[i];
        grav[i] = 10.0;
        met[i] = 0.0;
        mu_s[i] = 1.0 / sqrt(3.0);
        Tirr[i] = 1000.0;
        Tint[i] = 500.0;
        F0[i] = StBC * pow(Tirr[i], 4);
        Fint[i] = StBC * pow(Tint[i], 4);
        k_IR[i] = 1e-3;
        k_V[i] = 6e-4 * sqrt(Tirr[i] / 2000.0);
        gam[i] = k_V[i] / k_IR[i];
    }


    //std::fstream myfile("sig_hyb_HJ_53_a.txt", std::ios_base::in);

    //FILE *file;
    //file = fopen("sig_hyb_HJ_53_a.txt", "r");
    //std::ifstream input( "sig_hyb_HJ_53_a.txt" );


    //int x = 0;
    int i = 0;
    double a[ncol * nlay1];
    double b[ncol * nlay1];
    double pe[ncol * nlay1];
    double pu[ncol];
    double pl[ncol * nlay];
    double k_V_l[3 * ncol * nlay];
    double k_IR_l[2 * ncol * nlay];
    double k_V_l_1D[ncol * nlay];
    double k_IR_l_1D[ncol * nlay];
    double T[ncol * nlay];
    double prc[ncol];
    double Teff[ncol];
    double AB[ncol];
    double gam_V[ncol * 3];
    double Beta_V[ncol * 3];
    double Beta[ncol * 2];
    double gam_1[ncol];
    double gam_2[ncol];
    double gam_P[ncol];
    double tau_lim[ncol];

    //double t_tot = 0.0;
    //int inan = 0;
    //int k=0;
    //int n=0;
    double dT_rad[ncol * nlay];
    double dT_conv[ncol * nlay];
    double net_F[ncol * nlay1];

    //double seconds=0;
    time_t timer1, timer2;

    std::ofstream myfile;

    // Read in sigma hybrid grid values
    std::ifstream inFile;
    inFile.open(a_sh);
    if (!inFile)
    {
        //cout << "\nError opening the file: " << a_sh << endl;
        return 13;
    }
    for (i = 0; i < nlay1; i++)
    {
        inFile >> a[i];
    }
    inFile.close();

    inFile.open(b_sh);
    if (!inFile)
    {
        //cout << "\nError opening the file: " << b_sh << endl;
        return 13;
    }
    for (i = 0; i < nlay1; i++)
    {
        inFile >> b[i];
    }
    inFile.close();

    cout.precision(17);


    // Contruct pressure array in pa

    for (int c = 0; c < ncol; c++)
    {
        for (i = 0; i < nlay1; i++)
        {
            pe[c * nlay1 + i] = a[c * c + i] + b[c * nlay1 + i] * p0;
           // cout << "pe[i]  " << pe[c * nlay1 + i] << endl;

        }

        pu[c] = pe[c * nlay1 + 0];

    }





    //! Pressure layers

    for (int c = 0; c < ncol; c++)
    {
        for (i = 0; i < nlay; i++)
        {
            pl[c * nlay + i] = (pe[c * nlay + i + 1] - pe[c * nlay + i]) / (logl(pe[c * nlay + i + 1]) - logl(pe[c * nlay + i]));

        }
    }



    /*
    cout << "Tint | Tirr | p0 | pu | mu_s | grav " << endl;
    cout << Tint[0] << " | " << Tirr[0] << " | " <<
        p0 / 1e5 << " | " << pu[0] / 1e5 << " | " <<
        mu_s[0] << " | " << grav[0] << endl;
    cout << "-------------------------------" << endl;
    */

    // Semi-grey atmosphere values (here they are not used, but just need to be passed to IC routine)

    for (int c = 0; c < ncol; c++)
    {
        for (i = 0; i < nlay; i++)
        {
            k_V_l[c * nlay + 0 + i] = k_V[c];
            k_IR_l[c * nlay + 0 + i] = k_IR[c];
            k_V_l_1D[c * nlay + i] = k_V[c];
            k_IR_l_1D[c * nlay + 0 + i] = k_IR[c];
        }
    }

    double fl = (double)1.0;

    double tau_hf_e[nlay1] = { 0 };
    double kRoss_hf_e[nlay1];
    double tau_IRl_hf_l[0 + nlay];
    double gradrad_hf_l[0 + nlay] = { 0.0 }, gradad_hf_l[0 + nlay] = { 0.0 };


    double work_pl[nlay];
    double work_pe[nlay1];
    double work_T[nlay];

    double work_gam_V[3];
    double work_Beta_V[3];
    double work_Beta[2];



    for (int c = 0; c < ncol; c++)
    {
        for (int i = 0; i < nlay; i++)
        {
            work_pl[i] = pl[c * nlay + i];
            work_T[i] = T[c * nlay + i];
        }
        for (int i = 0; i < nlay1; i++)
        {
            work_pe[i] = pe[c * nlay1 + i];
        }

        //  Parmentier IC 
        IC_profile(iIC, corr, nlay,
            p0, work_pl, work_pe, k_V_l_1D, k_IR_l_1D, Tint[c],
            mu_s[c], Tirr[c], grav[c], fl,
            work_T, prc[c], table_num, met[c], tau_hf_e, kRoss_hf_e, tau_IRl_hf_l, gradrad_hf_l, gradad_hf_l);

        // Parmentier opacity profile parameters - first get Bond albedo
        Teff[c] = powl((powl(Tint[c], 4) + (1.0 / sqrtl((double)3.0)) *
            powl(Tirr[c], 4)), 0.25);
        Bond_Parmentier(Teff[c], grav[c], AB[c]);

        // Recalculate Teff and then find parameters
        Teff[c] = powl((powl(Tint[c], 4) + (((double)1.0) - AB[c]) * mu_s[c] *
            powl(Tirr[c], 4)), (0.25));

        gam_Parmentier(Teff[c], table_num, work_gam_V,
            work_Beta_V, work_Beta, gam_1[c], gam_2[c], gam_P[c], tau_lim[c]);

        for (int i = 0; i < 3; i++)
        {
            gam_V[c * 3 + i] = work_gam_V[i];
            Beta_V[c * 3 + i] = work_Beta_V[i];
        }
        for (int i = 0; i < 2; i++)
        {
            Beta[c * 2 + i] = work_Beta[i];
        }
    }


    /*
    // Print variables from Parmentier non-grey scheme
    cout << "Teff | AB | gam_V [0,1,2] | Beta_V [0,1,2] | Beta[0,1] | gam_1 | gam_2 | gam_P | tau_lim | prc" << endl;
    cout << Teff[0] << " | " << AB[0] << " | " <<
        gam_V[0] << " | " << Beta_V[0] << " | " <<
        Beta[0] << " | " << gam_1 << " | " <<
        gam_2 << " | " << gam_P << " | " <<
        tau_lim << " | " << prc[0] / 1e5 << endl;
    cout << "   " << " | " << "   " << " | " <<
        gam_V[1] << " | " << Beta_V[1] << " | " <<
        Beta[1] << " | " << "   " << " | " <<
        "   " << " | " << "   " << " | " <<
        "   " << " | " << "   " << endl;
    cout << "   " << " | " << "   " << " | " <<
        gam_V[2] << " | " << Beta_V[2] << " | " <<
        "   " << " | " << "   " << " | " <<
        "   " << " | " << "   " << " | " <<
        "   " << " | " << "   " << endl;
    cout << "-------------------------------" << endl;
    */

    // Print T-p profile
    // Write out initial conditions
    myfile.open("FMS_RC_ic.out");

    for (i = 0; i < nlay; i++)
    {
        //cout << i << " | " << pl[i] / 1e5 << " | " << T[i] << endl;
        myfile << i << " " << pl[i] / 1e5 << " " << T[i] << endl;
    }
    myfile.close();


    // Time stepping loop
    //cout << "Start timestepping" << endl;
    ////  time code here
    time(&timer2);





    //////////////////////////////////////////////////////////////////////////

    // Initialize device parameters
    /*
    size_t double3;
    double3 = 3 * sizeof(float);
    double *host_k_IR_l, *host_k_V_l;
    host_k_IR_l = (double*)malloc(double3);
    host_k_V_l = (double*)malloc(double3);


    for (int i = 0; i < nlay; i++)
    {
        for (int k = 0; k < 2; k++)
        {
            host_k_IR_l[2 * nlay] = k_IR_l[k][i];
        }
    }

    for (int i = 0; i < nlay; i++)
    {
        for (int k = 0; k < 3; k++)
        {
            host_k_V_l[3 * nlay] = k_V_l[k][i];
        }
    }

    */






    
    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda( T,
        pl,
        pe,
        met,
        gam_V,
        gam_1,
        gam_2,
        Beta_V,
        Beta,
        net_F,
        mu_s,
        F0,
        Fint,
        grav,
        AB,
        cp_air,
        kappa_air,
        t_step,
        n_step,
        k_IR_l,
        k_V_l,
        dT_rad,
        dT_conv,
        nlay,
        ncol);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    








    ///////////////////////////////////////////////////////////////////////////////



        // cpu time end
    time(&timer1);




    // Output

    /*
    cout << "sec: | " << "hours: | " << "days: | " << endl;

    cout << t_tot << " | " << t_tot / 60.0 / 60.0 << " | " <<
        t_tot / 60.0 / 60.0 / 24.0 << endl;

    */
    /*
    // !write (iname,frmat) int(t_tot/60.0_dp/60.0_dp/24.0_dp)
    //open(newunit=u,file='FMS_RC_pp.out',action='readwrite')
    myfile.open("FMS_RC_pp.out");

    for (i = 0; i <  nlay; i++)
    {

        myfile << i << " | " << pl[i] << " | " << T[i] << " | " <<
            dT_rad[i] << " | " << dT_conv[i] << " | " << k_V_l[0][i] <<
            " | " << k_V_l[1][i] << " | " << k_V_l[2][i] << " | " <<
            k_IR_l[0][i] << " | " << k_IR_l[1][i] << endl;
    }
    myfile.close();


    cout <<  Tint << " | " <<  Tirr << " | " <<
         p0 << " | " << pu << " | " <<  mu_s << " | " <<
         gam << endl;

    */
    // print time difference <<<<<<<
    double seconds;
    seconds = difftime(timer1, timer2);
    cout <<  n_step << " | " << "took: " << seconds << endl;

    


    
    /// ///////////////////////////////////////////////////////////////////////////////////////////
    /// ///////////////////////////////////////////////////////////////////////////////////////////////
    

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.


hipError_t addWithCuda(
    double* T,
    double* pl,
    double* pe,
    double* met,
    double* gam_V,
    double* gam_1,
    double* gam_2,
    double* Beta_V,
    double* Beta,
    double* net_F,
    double* mu_s,
    double* F0,
    double* Fint,
    double* grav,
    double* AB,
    double* cp_air,
    double* kappa_air,
    double* t_step,
    int* n_step,
    double* host_k_IR_l,
    double* host_k_V_l,
    double* dT_rad,
    double* dT_conv,
    unsigned int nlay,
    unsigned int ncol) {

    hipError_t cudaStatus;



    // Initialize device parameters

    unsigned int* dev_nlay;
    unsigned int* dev_ncol;
    //const int *dev_nlay1 ;
    double* dev_dT_rad;
    double* dev_dT_conv;
    double* dev_T;            /// parallel parameter
    double* dev_pl;
    double* dev_pe;
    double* dev_met;
    double* dev_k_IR_l;
    double* dev_k_V_l;
    double* dev_gam_V;
    double* dev_gam_1;
    double* dev_gam_2;
    double* dev_Beta_V;
    double* dev_Beta;
    double* dev_net_F;  /// parallel parameter
    double* dev_mu_s;
    double* dev_F0;
    double* dev_Fint;
    double* dev_grav;
    double* dev_AB;
    double* dev_cp_air;
    double* dev_kappa_air;
    double* dev_t_step;
    int* dev_n_step;
    //int* dev_num;




    //Kitzman working variables nlay1
    double* tau_Ve__df_e, * tau_IRe__df_e, * Te__df_e, * be__df_e,
        * sw_down__df_e, * sw_down_b__df_e, * sw_up__df_e,
        * lw_down__df_e, * lw_down_b__df_e,
        * lw_up__df_e, * lw_up_b__df_e,
        * lw_net__df_e, * sw_net__df_e,

        // lw_grey_updown_linear working variables nlay
        * dtau__dff_l, * del__dff_l,
        * edel__dff_l, * e0i__dff_l, * e1i__dff_l,
        * Am__dff_l, * Bm__dff_l,
        * lw_up_g__dff_l, * lw_down_g__dff_l,

        // dry_adj_Ray working variables nlay
        * Tl_cc__df_l, * d_p__df_l;




    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }


    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_nlay, sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&dev_ncol, sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_dT_rad, ncol * nlay * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_dT_conv, (ncol * nlay * sizeof(double)));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_T, (ncol * nlay * sizeof(double)));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_pl, (ncol * nlay * sizeof(double)));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_pe, (ncol * (1 + nlay) * sizeof(double)));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_dT_rad, ncol * (nlay * sizeof(double)));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_met, (ncol * sizeof(double)));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&dev_k_IR_l, 2 * nlay * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&dev_k_V_l, 3 * nlay * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&dev_gam_V, ncol * 3 * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&dev_gam_1, ncol * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&dev_gam_2, ncol * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&dev_Beta_V, ncol * 3 * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&dev_Beta, ncol * 2 * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&dev_net_F, ncol * (1 + nlay) * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&dev_mu_s, ncol * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&dev_F0, ncol * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&dev_Fint, ncol * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&dev_grav, ncol * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&dev_AB, ncol * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&dev_cp_air, ncol * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&dev_kappa_air, ncol * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&dev_t_step, ncol * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&dev_n_step, ncol * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    //Kitzman working variables
    cudaStatus = hipMalloc((void**)&tau_Ve__df_e, ncol * (nlay + 1) * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&tau_IRe__df_e, ncol * (nlay + 1) * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&Te__df_e, ncol * (nlay + 1) * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&be__df_e, ncol * (nlay + 1) * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&sw_down__df_e, ncol * (nlay + 1) * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&sw_down_b__df_e, ncol * (nlay + 1) * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&sw_up__df_e, ncol * (nlay + 1) * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&lw_down__df_e, ncol * (nlay + 1) * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&lw_down_b__df_e, ncol * (nlay + 1) * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&lw_up__df_e, ncol * (nlay + 1) * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&lw_up_b__df_e, ncol * (nlay + 1) * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&lw_net__df_e, ncol * (nlay + 1) * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&sw_net__df_e, ncol * (nlay + 1) * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    // lw_grey_updown_linear working variables
    cudaStatus = hipMalloc((void**)&dtau__dff_l, ncol * nlay * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&del__dff_l, ncol * nlay * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&edel__dff_l, ncol * nlay * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&e0i__dff_l, ncol * nlay * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&e1i__dff_l, ncol * nlay * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&Am__dff_l, ncol * nlay * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&Bm__dff_l, ncol * nlay * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&lw_up_g__dff_l, ncol * nlay * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&lw_down_g__dff_l, ncol * nlay * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    // dry_adj_Ray
    cudaStatus = hipMalloc((void**)&Tl_cc__df_l, ncol * nlay * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&d_p__df_l, ncol * nlay * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    // Allocate memory on the device

    //hipMalloc(&dev_nlay, sizeof(int));
    //hipMalloc(&dev_nlay1, sizeof(const int));
    //hipMalloc((void**)&dev_dT_rad, nlay * sizeof(double));
    //hipMalloc((void**)&dev_dT_conv, nlay * sizeof(double));
    //hipMalloc((void**)&dev_T, nlay * sizeof(double));
    //hipMalloc((void**)&dev_pl, nlay * sizeof(double));
    //hipMalloc((void**)&dev_pe, nlay1 * sizeof(double));
    //hipMalloc(&dev_met, sizeof(double));
    //hipMalloc((void**)&dev_k_IR_l, 2 * nlay * sizeof(double));
    //hipMalloc((void**)&dev_k_V_l, 3 * nlay * sizeof(double));
    //hipMalloc((void**)&dev_gam_V, 3 * sizeof(double));
    //hipMalloc((void**)&dev_gam_1, sizeof(double));
    //hipMalloc((void**)&dev_gam_2, sizeof(double));
    //hipMalloc((void**)&dev_Beta_V, 3 * sizeof(double));
    //hipMalloc((void**)&dev_Beta, 2 * sizeof(double));
    //hipMalloc((void**)&dev_net_F, (1+nlay) * sizeof(double));
    //hipMalloc((void**)&dev_mu_s, sizeof(double));
    //hipMalloc((void**)&dev_F0, sizeof(double));
    //hipMalloc((void**)&dev_Fint, sizeof(double));
    //hipMalloc((void**)&dev_grav, sizeof(double));
    //hipMalloc((void**)&dev_AB, sizeof(double));
    //hipMalloc((void**)&dev_cp_air, sizeof(double));
    //hipMalloc((void**)&dev_kappa_air, sizeof(double));
    //hipMalloc((void**)&dev_t_step, sizeof(int));
    //hipMalloc((void**)&dev_n_step, sizeof(int));
    //hipMalloc((void**)&dev_num, sizeof(int));




    // Copy data from the host to the device (CPU -> GPU)

    cudaStatus = hipMemcpy(dev_nlay, &nlay, sizeof(const int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    //hipMemcpy(dev_nlay1,  nlay,  sizeof(const int), hipMemcpyHostToDevice);
    //hipMemcpy(dev_dT_rad,  nlay,  nlay * sizeof(int), hipMemcpyHostToDevice);
    //hipMemcpy(dev_dT_conv,  nlay,  nlay * sizeof(int), hipMemcpyHostToDevice);
    cudaStatus = hipMemcpy(dev_T, T, nlay * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(dev_pl, pl, nlay * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(dev_pe, pe, (nlay + 1) * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(dev_met, met, sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(dev_k_IR_l, host_k_IR_l, 2 * nlay * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(dev_k_V_l, host_k_V_l, 3 * nlay * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(dev_gam_V, gam_V, 3 * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(dev_gam_1, gam_1, sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(dev_gam_2, gam_2, sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(dev_Beta_V, Beta_V, 3 * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(dev_Beta, Beta, 2 * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(dev_net_F, net_F, (nlay + 1) * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(dev_mu_s, mu_s, sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(dev_F0, F0, sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(dev_Fint, Fint, sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(dev_grav, grav, sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(dev_AB, AB, sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(dev_cp_air, cp_air, sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(dev_kappa_air, kappa_air, sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(dev_t_step, t_step, sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(dev_n_step, n_step, sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }






    // Threads per CTA (1024)
    //dim3 NUM_THREADS = 256;   //1 << 10;
    //dim3 NUM_THREADS = 2;
    //const int NUM_THREADS = 2;

    // CTAs per Grid
    // We need to launch at LEAST as many threads as we have elements
    // This equation pads an extra CTA to the grid if N cannot evenly be divided
    // by NUM_THREADS (e.g. N = 1025, NUM_THREADS = 1024)
    //dim3 NUM_BLOCKS = 1;   // (N + NUM_THREADS - 1) / NUM_THREADS;

    //dim3 NBRT((2 / NUM_THREADS) + 1, 1, 1);
    //dim3 NB = 2;


    // Launch the kernel on the GPU
    /*
    kernel_RT_loop << <NBRT, NUM_THREADS >> > (
        dev_dT_rad,
        dev_dT_conv,
        dev_T,
        dev_pl,
        dev_pe,
        dev_met,
        dev_k_IR_l,
        dev_k_V_l,
        dev_gam_V,
        dev_gam_1,
        dev_gam_2,
        dev_Beta_V,
        dev_Beta,
        dev_net_F,
        dev_mu_s,
        dev_F0,
        dev_Fint,
        dev_grav,
        dev_AB,
        dev_cp_air,
        dev_kappa_air,
        dev_t_step,
        dev_n_step,
        0,
        nlay,

        tau_Ve__df_e, tau_IRe__df_e, Te__df_e, be__df_e, //Kitzman working variables
        sw_down__df_e, sw_down_b__df_e, sw_up__df_e,
        lw_down__df_e, lw_down_b__df_e,
        lw_up__df_e, lw_up_b__df_e,
        lw_net__df_e, sw_net__df_e,

        dtau__dff_l, del__dff_l, // lw_grey_updown_linear working variables
        edel__dff_l, e0i__dff_l, e1i__dff_l,
        Am__dff_l, Bm__dff_l,
        lw_up_g__dff_l, lw_down_g__dff_l,

        Tl_cc__df_l, d_p__df_l); // dry_adj_Ray working variables
        */


                // Check for any errors launching the kernel
            cudaStatus = hipGetLastError();
            if (cudaStatus != hipSuccess) {
                fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
                goto Error;
            }

            // hipDeviceSynchronize waits for the kernel to finish, and returns
            // any errors encountered during the launch.
            cudaStatus = hipDeviceSynchronize();
            if (cudaStatus != hipSuccess) {
                fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
                goto Error;
            }







            // Copy output vector from GPU buffer to host memory.

            cudaStatus = hipMemcpy(dT_rad, dev_dT_rad, nlay * sizeof(double), hipMemcpyDeviceToHost);
            if (cudaStatus != hipSuccess) {
                fprintf(stderr, "hipMalloc failed!");
                goto Error;
            }
            cudaStatus = hipMemcpy(dT_conv, dev_dT_conv, nlay * sizeof(double), hipMemcpyDeviceToHost);
            if (cudaStatus != hipSuccess) {
                fprintf(stderr, "hipMalloc failed!");
                goto Error;
            }
            cudaStatus = hipMemcpy(T, dev_T, nlay * sizeof(double), hipMemcpyDeviceToHost);
            if (cudaStatus != hipSuccess) {
                fprintf(stderr, "hipMalloc failed!");
                goto Error;
            }
            cudaStatus = hipMemcpy(host_k_IR_l, dev_k_IR_l, 2 * nlay * sizeof(int), hipMemcpyDeviceToHost);
            if (cudaStatus != hipSuccess) {
                fprintf(stderr, "hipMalloc failed!");
                goto Error;
            }
            cudaStatus = hipMemcpy(host_k_V_l, dev_k_V_l, 3 * nlay * sizeof(int), hipMemcpyDeviceToHost);
            if (cudaStatus != hipSuccess) {
                fprintf(stderr, "hipMalloc failed!");
                goto Error;
            }
            cudaStatus = hipMemcpy(net_F, dev_net_F, (nlay + 1) * sizeof(double), hipMemcpyDeviceToHost);
            if (cudaStatus != hipSuccess) {
                fprintf(stderr, "hipMalloc failed!");
                goto Error;
            }


        Error:
            // Release GPU memory


            //hipFree(dev_nlay);
            //hipFree(dev_nlay1);
            hipFree(dev_dT_rad);
            hipFree(dev_dT_conv);
            hipFree(dev_T);
            hipFree(dev_pl);
            hipFree(dev_pe);
            hipFree(dev_met);
            hipFree(dev_k_IR_l);
            hipFree(dev_k_V_l);
            hipFree(dev_gam_V);
            hipFree(dev_gam_1);
            hipFree(dev_gam_2);
            hipFree(dev_Beta_V);
            hipFree(dev_Beta);
            hipFree(dev_net_F);
            hipFree(dev_mu_s);
            hipFree(dev_F0);
            hipFree(dev_Fint);
            hipFree(dev_grav);
            hipFree(dev_AB);
            hipFree(dev_cp_air);
            hipFree(dev_kappa_air);
            hipFree(dev_t_step);
            hipFree(dev_n_step);
            //hipFree(dev_num);

            hipFree(tau_Ve__df_e);
            hipFree(tau_IRe__df_e);
            hipFree(Te__df_e);
            hipFree(be__df_e);
            hipFree(sw_down__df_e);
            hipFree(sw_down_b__df_e);
            hipFree(sw_up__df_e);
            hipFree(lw_down__df_e);
            hipFree(lw_down_b__df_e);
            hipFree(lw_up__df_e);
            hipFree(lw_up_b__df_e);
            hipFree(lw_net__df_e);
            hipFree(sw_net__df_e);
            hipFree(dtau__dff_l);
            hipFree(del__dff_l);
            hipFree(edel__dff_l);
            hipFree(e0i__dff_l);
            hipFree(e1i__dff_l);
            hipFree(Am__dff_l);
            hipFree(Bm__dff_l);
            hipFree(lw_up_g__dff_l);
            hipFree(lw_down_g__dff_l);

            hipFree(Tl_cc__df_l);
            hipFree(d_p__df_l);

            return cudaStatus;

        }
