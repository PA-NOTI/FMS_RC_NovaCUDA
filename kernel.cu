#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <math.h>
#include <time.h>
#include <stdlib.h>
#include <fstream>
#include <time.h>
#include "IC_mod.h"
#include "picket_fence_cuda.h"


using namespace std;

hipError_t addWithCuda(
    double* T,
    double* pl,
    double* pe,
    double* met,
    double* gam_V,
    double* gam_1,
    double* gam_2,
    double* Beta_V,
    double* Beta,
    double* net_F,
    double* mu_s,
    double* F0,
    double* Fint,
    double* grav,
    double* AB,
    double* cp_air,
    double* kappa_air,
    double* t_step,
    double* n_step,
    double* host_k_IR_l,
    double* host_k_V_l,
    double* dT_rad,
    double* dT_conv,
    unsigned int nlay,
    unsigned int ncol
);


__global__ void kernel_RT_loop(
    //int nlay,
    //int nlay1,
    double* dT_rad,
    double* dT_conv,
    double* T,            /// parallel parameter
    double* pl,
    double* pe,
    double* met,
    double* k_IR_l,
    double* k_V_l,
    double* gam_V,
    double* gam_1,
    double* gam_2,
    double* Beta_V,
    double* Beta,
    double* net_F,  /// parallel parameter
    double* mu_s,
    double* F0,
    double* Fint,
    double* grav,
    double* AB,
    double* cp_air,
    double* kappa_air,
    double* t_step,
    double* n_step,
    int num,
    const int nlay,
    double* tau_Ve__df_e, double* tau_IRe__df_e, double* Te__df_e, double* be__df_e, //Kitzman working variables
    double* sw_down__df_e, double* sw_down_b__df_e, double* sw_up__df_e,
    double* lw_down__df_e, double* lw_down_b__df_e,
    double* lw_up__df_e, double* lw_up_b__df_e,
    double* lw_net__df_e, double* sw_net__df_e,

    double* dtau__dff_l, double* del__dff_l, // lw_grey_updown_linear working variables
    double* edel__dff_l, double* e0i__dff_l, double* e1i__dff_l,
    double* Am__dff_l, double* Bm__dff_l,
    double* lw_up_g__dff_l, double* lw_down_g__dff_l,

    double* Tl_cc__df_l, double* d_p__df_l //dry_adj_Ray working variables
)
{



    int id = blockIdx.x * blockDim.x + threadIdx.x;
    //double t_tot = 0.0;
    //int inan = 0;

    if (id < num) {

        for (int i = 0; i < n_step[id]; i++)
        {

            for (int k = 0; k < nlay; k++)
            {
                dT_rad[id * nlay + k] = 0.0;
                dT_conv[id * nlay + k] = 0.0;
            }

            // kappa calculation loop here if using non-constant kappa
            for (int level = 0; level < nlay; level++)
            {

                kernel_k_Ross_Freedman(T[id * nlay + level], pl[id * nlay + level], met[id], k_IR_l[id * nlay * 2 + 0 * nlay + level]);

                // Find the visual Rosseland mean opacity from gam_V


                for (int channel = 0; channel < 3; channel++)
                {
                    k_V_l[id * nlay * 3 + channel * nlay + level] = k_IR_l[id * nlay * 2 + 0 * nlay + level] * gam_V[id * 3 + channel];
                }


                // Find the IR Rosseland mean opacity in each IR picket fence band
                // Note: 2nd band done first here to avoid overwrite

                for (int channel = 0; channel < 2; channel++)
                {
                    k_IR_l[id * nlay * 2 + channel * nlay + level] = k_IR_l[id * nlay * 2 + 0 * nlay + level] * gam_2[id];
                    k_IR_l[id * nlay * 2 + channel * nlay + level] = k_IR_l[id * nlay * 2 + 0 * nlay + level] * gam_1[id];
                }



            }





            // !! Radiation - Comment in what scheme you want to use - Heng model won't work!
            //!call Heng_TS_noscatt(nlay, nlay1, T, Ts, pl, pe, tau_IRl, tau_IRe, tau_V, dT_rad, dT_s, mu, F0, Fint)
            //!call Toon_TS_noscatt(nlay, nlay1, T, pl, pe, k_V_l, k_IR_l, Beta_V, Beta, net_F, mu_s, F0, Fint, grav, AB)
            //!call DISORT_TS(nlay, nlay1, T, pl, pe, k_V_l, k_IR_l, Beta_V, Beta, net_F, mu_s, F0, Tint, grav, AB)
            Kitzmann_TS_noscatt(nlay,
                nlay + 1, T, pl, pe, k_V_l, k_IR_l, Beta_V, Beta, net_F,
                mu_s[0], F0[0], Fint[0], grav[0], AB[0],

                tau_Ve__df_e, tau_IRe__df_e, Te__df_e, be__df_e, //Kitzman working variables
                sw_down__df_e, sw_down_b__df_e, sw_up__df_e,
                lw_down__df_e, lw_down_b__df_e,
                lw_up__df_e, lw_up_b__df_e,
                lw_net__df_e, sw_net__df_e,

                dtau__dff_l, del__dff_l, // lw_grey_updown_linear working variables
                edel__dff_l, e0i__dff_l, e1i__dff_l,
                Am__dff_l, Bm__dff_l,
                lw_up_g__dff_l, lw_down_g__dff_l);

            for (int level = 0; level < nlay; level++)
            {
                dT_rad[level] = (grav[0] / cp_air[0]) *
                    (net_F[level + 1] - net_F[level]) / (pe[level + 1] - pe[level]);

            }



            // Dry convective adjustment using Ray's code
            Ray_dry_adj(nlay, nlay + 1,
                t_step[0], kappa_air[0], T, pl, pe, dT_conv,
                Tl_cc__df_l, d_p__df_l);

            // Forward march the temperature change from convection
            for (int level = 0; level < nlay; level++)
            {
                T[level] = T[level] + t_step[0] * (dT_conv[level] + dT_rad[level]);
            }


            for (int k = 0; k < nlay; k++)
            {

                /*

                if (isnan(T[k]) == true)
                {


                    for (int n = 0; n < nlay; n++)
                    {

                        cout << " k = " << k << endl;
                        cout << n << " | " << T[n] << " | " << net_F[n] << " | " <<
                            dT_rad[n] << " | " << dT_conv[n] << endl;
                    }


                    cout <<  nlay1 << " | " <<
                        net_F[ nlay1] << endl;
                    */
                    /*
                            inan = 1;
                            break;
                        }
                    }
                    if (inan == 1)
                    {
                        break;
                    }

                    //t_tot = t_tot + t_step;



                    */
            }


        }




    }
}

                


int main()
{
    

    const double StBC = 5.670374419e-8;


    // Number of columns, layers and edges
    const int ncol = 2;
    const int nlay = 52;
    const int nlay1 = nlay + 1;



    const std::string frmat = "(I6.6)";

    // Read in sigma hybrid grid values
    const std::string a_sh = "sig_hyb_HJ_53_a.txt";
    const std::string b_sh = "sig_hyb_HJ_53_b.txt";

    // Surface pressure (pa)
    const double p0 = 2.2e7;

    // step in seconds
    double t_step[ncol] = { 30.0 };

    // Number of steps
    double n_step[ncol] = { 1e5 };

    // Constants
    //double R = 8.31446261815324; //     ! Ideal gas constant
    double Rd_air[ncol]; //          ! Specific gas constant
    double cp_air[ncol]; //           ! Rd_air/kappa_air ! Heat capacity of air
    double kappa_air[ncol];// ! kappa = Rd/cp
    double grav[ncol]; //              ! Gravity
    double met[ncol];   //                ! Metallicity in dex solar, solar[M/H] = 0.0


    //! short wave Zenith angle
    double mu_s[ncol];

    double Tirr[ncol]; //               ! Irradiation temperature
    double Tint[ncol]; //                 ! Internal temperature

    double F0[ncol];   //        ! Substellar point irradiation flux

    double Fint[ncol];//      ! Internal flux

    double k_IR[ncol];        //           ! Constant IR opacity
    double k_V[ncol];//! Constant V opacity

    double gam[ncol]; //               ! Gamma ratio

    // Parmentier IC and parameters
    int iIC = 4; //       ! IC choice here
    bool corr = true; //  ! Do adibatic correction

    int table_num = 1; // ! Table 1 = with TiO/VO, Table 2 = without TiO/VO

    for (size_t i = 0; i < ncol; i++)
    {
        Rd_air[i] = 3556.8;
        cp_air[i] = 1.3e4;
        kappa_air[i] = Rd_air[i] / cp_air[i];
        grav[i] = 10.0;
        met[i] = 0.0;
        mu_s[i] = 1.0 / sqrt(3.0);
        Tirr[i] = 1000.0;
        Tint[i] = 500.0;
        F0[i] = StBC * pow(Tirr[i], 4);
        Fint[i] = StBC * pow(Tint[i], 4);
        k_IR[i] = 1e-3;
        k_V[i] = 6e-4 * sqrt(Tirr[i] / 2000.0);
        gam[i] = k_V[i] / k_IR[i];
    }


    //std::fstream myfile("sig_hyb_HJ_53_a.txt", std::ios_base::in);

    //FILE *file;
    //file = fopen("sig_hyb_HJ_53_a.txt", "r");
    //std::ifstream input( "sig_hyb_HJ_53_a.txt" );


    //int x = 0;
    int i = 0;
    double a[ncol * nlay1];
    double b[ncol * nlay1];
    double pe[ncol * nlay1];
    double pu[ncol];
    double pl[ncol * nlay];
    double k_V_l[3 * ncol * nlay];
    double k_IR_l[2 * ncol * nlay];
    double k_V_l_1D[ncol * nlay];
    double k_IR_l_1D[ncol * nlay];
    double T[ncol * nlay];
    double prc[ncol];
    double Teff[ncol];
    double AB[ncol];
    double gam_V[ncol * 3];
    double Beta_V[ncol * 3];
    double Beta[ncol * 2];
    double gam_1[ncol];
    double gam_2[ncol];
    double gam_P[ncol];
    double tau_lim[ncol];

    //double t_tot = 0.0;
    //int inan = 0;
    //int k=0;
    //int n=0;
    double dT_rad[ncol * nlay];
    double dT_conv[ncol * nlay];
    double net_F[ncol * nlay1];

    //double seconds=0;
    time_t timer1, timer2;

    std::ofstream myfile;

    // Read in sigma hybrid grid values
    std::ifstream inFile;
    inFile.open(a_sh);
    if (!inFile)
    {
        //cout << "\nError opening the file: " << a_sh << endl;
        return 13;
    }
    for (i = 0; i < nlay1; i++)
    {
        inFile >> a[i];
    }
    inFile.close();

    inFile.open(b_sh);
    if (!inFile)
    {
        //cout << "\nError opening the file: " << b_sh << endl;
        return 13;
    }
    for (i = 0; i < nlay1; i++)
    {
        inFile >> b[i];
    }
    inFile.close();

    cout.precision(17);


    // Contruct pressure array in pa

    for (int c = 0; c < ncol; c++)
    {
        for (i = 0; i < nlay1; i++)
        {
            pe[c * nlay1 + i] = a[c * c + i] + b[c * nlay1 + i] * p0;
           // cout << "pe[i]  " << pe[c * nlay1 + i] << endl;

        }

        pu[c] = pe[c * nlay1 + 0];

    }





    //! Pressure layers

    for (int c = 0; c < ncol; c++)
    {
        for (i = 0; i < nlay; i++)
        {
            pl[c * nlay + i] = (pe[c * nlay + i + 1] - pe[c * nlay + i]) / (logl(pe[c * nlay + i + 1]) - logl(pe[c * nlay + i]));

        }
    }



    /*
    cout << "Tint | Tirr | p0 | pu | mu_s | grav " << endl;
    cout << Tint[0] << " | " << Tirr[0] << " | " <<
        p0 / 1e5 << " | " << pu[0] / 1e5 << " | " <<
        mu_s[0] << " | " << grav[0] << endl;
    cout << "-------------------------------" << endl;
    */

    // Semi-grey atmosphere values (here they are not used, but just need to be passed to IC routine)

    for (int c = 0; c < ncol; c++)
    {
        for (i = 0; i < nlay; i++)

        {
            k_V_l[c * nlay*3 + 0 + i] = k_V[c];
            k_IR_l[c * nlay*2 + 0 + i] = k_IR[c];

        }
        k_V_l_1D[c * nlay] = k_V[c];
        k_IR_l_1D[c * nlay] = k_IR[c];
    }

    double fl = (double)1.0;

    double tau_hf_e[nlay1] = { 0 };
    double kRoss_hf_e[nlay1];
    double tau_IRl_hf_l[0 + nlay];
    double gradrad_hf_l[0 + nlay] = { 0.0 }, gradad_hf_l[0 + nlay] = { 0.0 };


    double work_pl[nlay];
    double work_pe[nlay1];
    double work_T[nlay];

    double work_gam_V[3];
    double work_Beta_V[3];
    double work_Beta[2];



    for (int c = 0; c < ncol; c++)
    {
        for (int i = 0; i < nlay; i++)
        {
            work_pl[i] = pl[c * nlay + i];
            work_T[i] = T[c * nlay + i];
        }
        for (int i = 0; i < nlay1; i++)
        {
            work_pe[i] = pe[c * nlay1 + i];
        }

        //  Parmentier IC 
        IC_profile(iIC, corr, nlay,
            p0, work_pl, work_pe, k_V_l_1D, k_IR_l_1D, Tint[c],
            mu_s[c], Tirr[c], grav[c], fl,
            work_T, prc[c], table_num, met[c], tau_hf_e, kRoss_hf_e, tau_IRl_hf_l, gradrad_hf_l, gradad_hf_l);

        // Parmentier opacity profile parameters - first get Bond albedo
        Teff[c] = powl((powl(Tint[c], 4) + (1.0 / sqrtl((double)3.0)) *
            powl(Tirr[c], 4)), 0.25);
        Bond_Parmentier(Teff[c], grav[c], AB[c]);

        // Recalculate Teff and then find parameters
        Teff[c] = powl((powl(Tint[c], 4) + (((double)1.0) - AB[c]) * mu_s[c] *
            powl(Tirr[c], 4)), (0.25));

        gam_Parmentier(Teff[c], table_num, work_gam_V,
            work_Beta_V, work_Beta, gam_1[c], gam_2[c], gam_P[c], tau_lim[c]);

        for (int i = 0; i < 3; i++)
        {
            gam_V[c * 3 + i] = work_gam_V[i];
            Beta_V[c * 3 + i] = work_Beta_V[i];
        }
        for (int i = 0; i < 2; i++)
        {
            Beta[c * 2 + i] = work_Beta[i];
        }
        for (int i = 0; i < nlay; i++)
        {
             pl[c * nlay + i] = work_pl[i];
            T[c * nlay + i] = work_T[i];
        }
        for (int i = 0; i < nlay1; i++)
        {
            pe[c * nlay1 + i] = work_pe[i];
        }
    }


    /*
    // Print variables from Parmentier non-grey scheme
    cout << "Teff | AB | gam_V [0,1,2] | Beta_V [0,1,2] | Beta[0,1] | gam_1 | gam_2 | gam_P | tau_lim | prc" << endl;
    cout << Teff[0] << " | " << AB[0] << " | " <<
        gam_V[0] << " | " << Beta_V[0] << " | " <<
        Beta[0] << " | " << gam_1 << " | " <<
        gam_2 << " | " << gam_P << " | " <<
        tau_lim << " | " << prc[0] / 1e5 << endl;
    cout << "   " << " | " << "   " << " | " <<
        gam_V[1] << " | " << Beta_V[1] << " | " <<
        Beta[1] << " | " << "   " << " | " <<
        "   " << " | " << "   " << " | " <<
        "   " << " | " << "   " << endl;
    cout << "   " << " | " << "   " << " | " <<
        gam_V[2] << " | " << Beta_V[2] << " | " <<
        "   " << " | " << "   " << " | " <<
        "   " << " | " << "   " << " | " <<
        "   " << " | " << "   " << endl;
    cout << "-------------------------------" << endl;
    */

    // Print T-p profile
    // Write out initial conditions
    myfile.open("FMS_RC_ic.out");

    for (i = 0; i < nlay; i++)
    {
        //cout << i << " | " << pl[i] / 1e5 << " | " << T[i] << endl;
        myfile << i << " " << pl[i] / 1e5 << " " << T[i] << endl;
    }
    myfile.close();


    // Time stepping loop
    //cout << "Start timestepping" << endl;
    ////  time code here
    time(&timer2);

    //////////////////////////////////////////////////////////////////////////

        
    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda( T,
        pl,
        pe,
        met,
        gam_V,
        gam_1,
        gam_2,
        Beta_V,
        Beta,
        net_F,
        mu_s,
        F0,
        Fint,
        grav,
        AB,
        cp_air,
        kappa_air,
        t_step,
        n_step,
        k_IR_l,
        k_V_l,
        dT_rad,
        dT_conv,
        nlay,
        ncol);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    








    ///////////////////////////////////////////////////////////////////////////////



        // cpu time end
    time(&timer1);




    // Output

    /*
    cout << "sec: | " << "hours: | " << "days: | " << endl;

    cout << t_tot << " | " << t_tot / 60.0 / 60.0 << " | " <<
        t_tot / 60.0 / 60.0 / 24.0 << endl;

    */
    
    // !write (iname,frmat) int(t_tot/60.0_dp/60.0_dp/24.0_dp)
    //open(newunit=u,file='FMS_RC_pp.out',action='readwrite')
    myfile.open("FMS_RC_pp.out");

    for (i = 0; i <  nlay; i++)
    {
        

        myfile << i << " | " << pl[i] << " | " << T[i] << " | " <<
            dT_rad[i] << " | " << dT_conv[i] << " | " << k_V_l[0 * ncol * nlay + 0 * nlay + i] <<
            " | " << k_V_l[0*ncol*nlay+1*nlay+i] << " | " << k_V_l[0 * ncol * nlay + 2 * nlay + i] << " | " <<
            k_IR_l[0 * ncol * nlay + 0 * nlay + i] << " | " << k_IR_l[0 * ncol * nlay + 1 * nlay + i] << endl;
    }
    myfile.close();

    /*
    cout <<  Tint << " | " <<  Tirr << " | " <<
         p0 << " | " << pu << " | " <<  mu_s << " | " <<
         gam << endl;

    */

    
    // print time difference <<<<<<<
    double seconds;
    seconds = difftime(timer1, timer2);
    cout <<  n_step << " | " << "took: " << seconds << endl;

    


    
    /// ///////////////////////////////////////////////////////////////////////////////////////////
    /// ///////////////////////////////////////////////////////////////////////////////////////////////
    

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.


hipError_t addWithCuda(
    double* T,
    double* pl,
    double* pe,
    double* met,
    double* gam_V,
    double* gam_1,
    double* gam_2,
    double* Beta_V,
    double* Beta,
    double* net_F,
    double* mu_s,
    double* F0,
    double* Fint,
    double* grav,
    double* AB,
    double* cp_air,
    double* kappa_air,
    double* t_step,
    double* n_step,
    double* host_k_IR_l,
    double* host_k_V_l,
    double* dT_rad,
    double* dT_conv,
    unsigned int nlay,
    unsigned int ncol) {

    hipError_t cudaStatus;



    // Initialize device parameters

    unsigned int* dev_nlay;
    unsigned int* dev_ncol;
    //const int *dev_nlay1 ;
    double* dev_dT_rad;
    double* dev_dT_conv;
    double* dev_T;            /// parallel parameter
    double* dev_pl;
    double* dev_pe;
    double* dev_met;
    double* dev_k_IR_l;
    double* dev_k_V_l;
    double* dev_gam_V;
    double* dev_gam_1;
    double* dev_gam_2;
    double* dev_Beta_V;
    double* dev_Beta;
    double* dev_net_F;  /// parallel parameter
    double* dev_mu_s;
    double* dev_F0;
    double* dev_Fint;
    double* dev_grav;
    double* dev_AB;
    double* dev_cp_air;
    double* dev_kappa_air;
    double* dev_t_step;
    double* dev_n_step;
    //int* dev_num;




    //Kitzman working variables nlay1
    double* tau_Ve__df_e, * tau_IRe__df_e, * Te__df_e, * be__df_e,
        * sw_down__df_e, * sw_down_b__df_e, * sw_up__df_e,
        * lw_down__df_e, * lw_down_b__df_e,
        * lw_up__df_e, * lw_up_b__df_e,
        * lw_net__df_e, * sw_net__df_e,

        // lw_grey_updown_linear working variables nlay
        * dtau__dff_l, * del__dff_l,
        * edel__dff_l, * e0i__dff_l, * e1i__dff_l,
        * Am__dff_l, * Bm__dff_l,
        * lw_up_g__dff_l, * lw_down_g__dff_l,

        // dry_adj_Ray working variables nlay
        * Tl_cc__df_l, * d_p__df_l;




    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }


    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_nlay, sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "dev_nlay hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&dev_ncol, sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "dev_ncol hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_dT_rad, ncol * nlay * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "dev_dT_rad hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_dT_conv, (ncol * nlay * sizeof(double)));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "dev_dT_conv hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_T, (ncol * nlay * sizeof(double)));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "dev_T hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_pl, (ncol * nlay * sizeof(double)));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "dev_pl hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_pe, (ncol * (1 + nlay) * sizeof(double)));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "dev_pe hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_dT_rad, ncol * (nlay * sizeof(double)));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "dev_dT_rad hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_met, (ncol * sizeof(double)));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "dev_met hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&dev_k_IR_l, 2 * nlay * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "dev_k_IR_l hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&dev_k_V_l, 3 * nlay * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "dev_k_V_l hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&dev_gam_V, ncol * 3 * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "dev_gam_V hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&dev_gam_1, ncol * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "dev_gam_1 hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&dev_gam_2, ncol * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "dev_gam_2 hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&dev_Beta_V, ncol * 3 * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "dev_Beta_V hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&dev_Beta, ncol * 2 * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "dev_Beta hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&dev_net_F, ncol * (1 + nlay) * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "dev_net_F hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&dev_mu_s, ncol * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "dev_mu_s hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&dev_F0, ncol * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "dev_F0 hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&dev_Fint, ncol * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "dev_Fint hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&dev_grav, ncol * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "dev_grav hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&dev_AB, ncol * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "dev_AB hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&dev_cp_air, ncol * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "dev_cp_air hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&dev_kappa_air, ncol * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "dev_kappa_air hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&dev_t_step, ncol * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "dev_t_step hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&dev_n_step, ncol * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "dev_n_step hipMalloc failed!");
        goto Error;
    }
    //Kitzman working variables
    cudaStatus = hipMalloc((void**)&tau_Ve__df_e, ncol * (nlay + 1) * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "tau_Ve__df_e hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&tau_IRe__df_e, ncol * (nlay + 1) * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "tau_IRe__df_e hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&Te__df_e, ncol * (nlay + 1) * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Te__df_e hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&be__df_e, ncol * (nlay + 1) * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "be__df_e hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&sw_down__df_e, ncol * (nlay + 1) * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "sw_down__df_e hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&sw_down_b__df_e, ncol * (nlay + 1) * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "sw_down_b__df_e hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&sw_up__df_e, ncol * (nlay + 1) * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "sw_up__df_e hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&lw_down__df_e, ncol * (nlay + 1) * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "lw_down__df_e hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&lw_down_b__df_e, ncol * (nlay + 1) * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "lw_down_b__df_e hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&lw_up__df_e, ncol * (nlay + 1) * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "lw_up__df_e hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&lw_up_b__df_e, ncol * (nlay + 1) * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "lw_up_b__df_e hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&lw_net__df_e, ncol * (nlay + 1) * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "lw_net__df_e hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&sw_net__df_e, ncol * (nlay + 1) * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "sw_net__df_e hipMalloc failed!");
        goto Error;
    }
    // lw_grey_updown_linear working variables
    cudaStatus = hipMalloc((void**)&dtau__dff_l, ncol * nlay * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "dtau__dff_l hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&del__dff_l, ncol * nlay * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "del__dff_l hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&edel__dff_l, ncol * nlay * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "edel__dff_l hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&e0i__dff_l, ncol * nlay * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "e0i__dff_l hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&e1i__dff_l, ncol * nlay * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "e1i__dff_l hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&Am__dff_l, ncol * nlay * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Am__dff_l hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&Bm__dff_l, ncol * nlay * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Bm__dff_l hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&lw_up_g__dff_l, ncol * nlay * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "lw_up_g__dff_l hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&lw_down_g__dff_l, ncol * nlay * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "lw_down_g__dff_l hipMalloc failed!");
        goto Error;
    }
    // dry_adj_Ray
    cudaStatus = hipMalloc((void**)&Tl_cc__df_l, ncol * nlay * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Tl_cc__df_l hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&d_p__df_l, ncol * nlay * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "d_p__df_l hipMalloc failed!");
        goto Error;
    }
    // Allocate memory on the device

    //hipMalloc(&dev_nlay, sizeof(int));
    //hipMalloc(&dev_nlay1, sizeof(const int));
    //hipMalloc((void**)&dev_dT_rad, nlay * sizeof(double));
    //hipMalloc((void**)&dev_dT_conv, nlay * sizeof(double));
    //hipMalloc((void**)&dev_T, nlay * sizeof(double));
    //hipMalloc((void**)&dev_pl, nlay * sizeof(double));
    //hipMalloc((void**)&dev_pe, nlay1 * sizeof(double));
    //hipMalloc(&dev_met, sizeof(double));
    //hipMalloc((void**)&dev_k_IR_l, 2 * nlay * sizeof(double));
    //hipMalloc((void**)&dev_k_V_l, 3 * nlay * sizeof(double));
    //hipMalloc((void**)&dev_gam_V, 3 * sizeof(double));
    //hipMalloc((void**)&dev_gam_1, sizeof(double));
    //hipMalloc((void**)&dev_gam_2, sizeof(double));
    //hipMalloc((void**)&dev_Beta_V, 3 * sizeof(double));
    //hipMalloc((void**)&dev_Beta, 2 * sizeof(double));
    //hipMalloc((void**)&dev_net_F, (1+nlay) * sizeof(double));
    //hipMalloc((void**)&dev_mu_s, sizeof(double));
    //hipMalloc((void**)&dev_F0, sizeof(double));
    //hipMalloc((void**)&dev_Fint, sizeof(double));
    //hipMalloc((void**)&dev_grav, sizeof(double));
    //hipMalloc((void**)&dev_AB, sizeof(double));
    //hipMalloc((void**)&dev_cp_air, sizeof(double));
    //hipMalloc((void**)&dev_kappa_air, sizeof(double));
    //hipMalloc((void**)&dev_t_step, sizeof(int));
    //hipMalloc((void**)&dev_n_step, sizeof(int));
    //hipMalloc((void**)&dev_num, sizeof(int));




    // Copy data from the host to the device (CPU -> GPU)

    cudaStatus = hipMemcpy(dev_nlay, &nlay, sizeof(const int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "dev_nlay hipMalloc failed!");
        goto Error;
    }
    //hipMemcpy(dev_nlay1,  nlay,  sizeof(const int), hipMemcpyHostToDevice);
    //hipMemcpy(dev_dT_rad,  nlay,  nlay * sizeof(int), hipMemcpyHostToDevice);
    //hipMemcpy(dev_dT_conv,  nlay,  nlay * sizeof(int), hipMemcpyHostToDevice);
    cudaStatus = hipMemcpy(dev_T, T, ncol* nlay * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "dev_T hipMemcpyHostToDevice failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(dev_pl, pl, ncol * nlay * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "dev_pl hipMemcpyHostToDevice failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(dev_pe, pe, ncol * (nlay + 1) * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "dev_pe hipMemcpyHostToDevice failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(dev_met, met, ncol * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "dev_met hipMemcpyHostToDevice failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(dev_k_IR_l, host_k_IR_l, ncol * 2 * nlay * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "dev_k_IR_l hipMemcpyHostToDevice failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(dev_k_V_l, host_k_V_l, ncol * 3 * nlay * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "dev_k_V_l hipMemcpyHostToDevice failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(dev_gam_V, gam_V, ncol * 3 * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "dev_gam_V hipMemcpyHostToDevice failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(dev_gam_1, gam_1, ncol * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "dev_gam_1 hipMemcpyHostToDevice failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(dev_gam_2, gam_2, ncol * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "dev_gam_2 hipMemcpyHostToDevice failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(dev_Beta_V, Beta_V, ncol * 3 * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "dev_Beta_V hipMemcpyHostToDevice failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(dev_Beta, Beta, ncol * 2 * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "dev_Beta hipMemcpyHostToDevice failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(dev_net_F, net_F, ncol * (nlay + 1) * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "dev_net_F hipMemcpyHostToDevice failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(dev_mu_s, mu_s, ncol * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "dev_mu_s hipMemcpyHostToDevice failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(dev_F0, F0, ncol * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "dev_F0 hipMemcpyHostToDevice failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(dev_Fint, Fint, ncol * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "dev_Fint hipMemcpyHostToDevice failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(dev_grav, grav, ncol * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "dev_grav hipMemcpyHostToDevice failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(dev_AB, AB, ncol * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "dev_AB hipMemcpyHostToDevice failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(dev_cp_air, cp_air, ncol * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "dev_cp_air hipMemcpyHostToDevice failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(dev_kappa_air, kappa_air, ncol * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "dev_kappa_air hipMemcpyHostToDevice failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(dev_t_step, t_step, ncol * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "dev_t_step hipMemcpyHostToDevice failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(dev_n_step, n_step, ncol * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "dev_n_step hipMemcpyHostToDevice failed!");
        goto Error;
    }






    // Threads per CTA (1024)
    //dim3 NUM_THREADS = 256;   //1 << 10;
    //dim3 NUM_THREADS = 2;
    //const int NUM_THREADS = 2;

    // CTAs per Grid
    // We need to launch at LEAST as many threads as we have elements
    // This equation pads an extra CTA to the grid if N cannot evenly be divided
    // by NUM_THREADS (e.g. N = 1025, NUM_THREADS = 1024)
    //dim3 NUM_BLOCKS = 1;   // (N + NUM_THREADS - 1) / NUM_THREADS;
    //dim3 NB = 2;
    //dim3 NBRT((2 / NB) + 1, 1, 1);
    


    // Launch the kernel on the GPU
    
    kernel_RT_loop << <1, 2 >> > (
        dev_dT_rad,
        dev_dT_conv,
        dev_T,
        dev_pl,
        dev_pe,
        dev_met,
        dev_k_IR_l,
        dev_k_V_l,
        dev_gam_V,
        dev_gam_1,
        dev_gam_2,
        dev_Beta_V,
        dev_Beta,
        dev_net_F,
        dev_mu_s,
        dev_F0,
        dev_Fint,
        dev_grav,
        dev_AB,
        dev_cp_air,
        dev_kappa_air,
        dev_t_step,
        dev_n_step,
        0,
        nlay,

        tau_Ve__df_e, tau_IRe__df_e, Te__df_e, be__df_e, //Kitzman working variables
        sw_down__df_e, sw_down_b__df_e, sw_up__df_e,
        lw_down__df_e, lw_down_b__df_e,
        lw_up__df_e, lw_up_b__df_e,
        lw_net__df_e, sw_net__df_e,

        dtau__dff_l, del__dff_l, // lw_grey_updown_linear working variables
        edel__dff_l, e0i__dff_l, e1i__dff_l,
        Am__dff_l, Bm__dff_l,
        lw_up_g__dff_l, lw_down_g__dff_l,

        Tl_cc__df_l, d_p__df_l); // dry_adj_Ray working variables
        


                // Check for any errors launching the kernel
            cudaStatus = hipGetLastError();
            if (cudaStatus != hipSuccess) {
                fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
                goto Error;
            }

            // hipDeviceSynchronize waits for the kernel to finish, and returns
            // any errors encountered during the launch.
            cudaStatus = hipDeviceSynchronize();
            if (cudaStatus != hipSuccess) {
                fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
                goto Error;
            }







            // Copy output vector from GPU buffer to host memory.

            cudaStatus = hipMemcpy(dT_rad, dev_dT_rad, ncol * nlay * sizeof(double), hipMemcpyDeviceToHost);
            if (cudaStatus != hipSuccess) {
                fprintf(stderr, "dev_dT_rad hipMemcpyDeviceToHost failed!");
                goto Error;
            }
            cudaStatus = hipMemcpy(dT_conv, dev_dT_conv, ncol * nlay * sizeof(double), hipMemcpyDeviceToHost);
            if (cudaStatus != hipSuccess) {
                fprintf(stderr, "dev_dT_conv hipMemcpyDeviceToHost failed!");
                goto Error;
            }
            cudaStatus = hipMemcpy(T, dev_T, ncol  * nlay * sizeof(double), hipMemcpyDeviceToHost);
            if (cudaStatus != hipSuccess) {
                fprintf(stderr, "dev_T hipMemcpyDeviceToHost failed!");
                goto Error;
            }
            cudaStatus = hipMemcpy(host_k_IR_l, dev_k_IR_l, ncol * 2 * nlay * sizeof(double), hipMemcpyDeviceToHost);
            if (cudaStatus != hipSuccess) {
                fprintf(stderr, "dev_k_IR_l hipMemcpyDeviceToHost failed!");
                goto Error;
            }
            cudaStatus = hipMemcpy(host_k_V_l, dev_k_V_l, ncol * 3 * nlay * sizeof(double), hipMemcpyDeviceToHost);
            if (cudaStatus != hipSuccess) {
                fprintf(stderr, "dev_k_V_l hipMemcpyDeviceToHost failed!");
                goto Error;
            }
            cudaStatus = hipMemcpy(net_F, dev_net_F, ncol * (nlay + 1) * sizeof(double), hipMemcpyDeviceToHost);
            if (cudaStatus != hipSuccess) {
                fprintf(stderr, "dev_net_F hipMemcpyDeviceToHost failed!");
                goto Error;
            }

        Error:
            // Release GPU memory


            //hipFree(dev_nlay);
            //hipFree(dev_nlay1);
            hipFree(dev_dT_rad);
            hipFree(dev_dT_conv);
            hipFree(dev_T);
            hipFree(dev_pl);
            hipFree(dev_pe);
            hipFree(dev_met);
            hipFree(dev_k_IR_l);
            hipFree(dev_k_V_l);
            hipFree(dev_gam_V);
            hipFree(dev_gam_1);
            hipFree(dev_gam_2);
            hipFree(dev_Beta_V);
            hipFree(dev_Beta);
            hipFree(dev_net_F);
            hipFree(dev_mu_s);
            hipFree(dev_F0);
            hipFree(dev_Fint);
            hipFree(dev_grav);
            hipFree(dev_AB);
            hipFree(dev_cp_air);
            hipFree(dev_kappa_air);
            hipFree(dev_t_step);
            hipFree(dev_n_step);
            //hipFree(dev_num);

            hipFree(tau_Ve__df_e);
            hipFree(tau_IRe__df_e);
            hipFree(Te__df_e);
            hipFree(be__df_e);
            hipFree(sw_down__df_e);
            hipFree(sw_down_b__df_e);
            hipFree(sw_up__df_e);
            hipFree(lw_down__df_e);
            hipFree(lw_down_b__df_e);
            hipFree(lw_up__df_e);
            hipFree(lw_up_b__df_e);
            hipFree(lw_net__df_e);
            hipFree(sw_net__df_e);
            hipFree(dtau__dff_l);
            hipFree(del__dff_l);
            hipFree(edel__dff_l);
            hipFree(e0i__dff_l);
            hipFree(e1i__dff_l);
            hipFree(Am__dff_l);
            hipFree(Bm__dff_l);
            hipFree(lw_up_g__dff_l);
            hipFree(lw_down_g__dff_l);

            hipFree(Tl_cc__df_l);
            hipFree(d_p__df_l);

            return cudaStatus;

        }
